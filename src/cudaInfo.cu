// Logger Lib
#define SPDLOG_ACTIVE_LEVEL SPDLOG_LEVEL_TRACE
#include <spdlog/spdlog.h>

#include "cudaInfo.cuh"

bool checkCuda() {
    int nDevices = 0;
    hipGetDeviceCount(&nDevices);
    if (nDevices == 0) {
        SPDLOG_ERROR("Error: checkCuda: No CUDA devices found");
        return false;
    }
    for (auto i = 0; i < nDevices; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        SPDLOG_INFO(spdlog::fmt_lib::format("Cuda Device {}", prop.name));
        SPDLOG_INFO(spdlog::fmt_lib::format("\t├ Compute Units {}", prop.multiProcessorCount));
        SPDLOG_INFO(spdlog::fmt_lib::format("\t├ Max Work Group Size {}", prop.warpSize));
        SPDLOG_INFO(spdlog::fmt_lib::format("\t├ Local Mem Size {}", prop.sharedMemPerBlock));
        SPDLOG_INFO(spdlog::fmt_lib::format("\t└ Global Mem Size {}", prop.totalGlobalMem));
    }
    return true;
}